#include "hip/hip_runtime.h"
#include <iostream>
#include "gpu_addVect.cuh"
#include "cuda_timer.hpp"

namespace AddVect{

    std::size_t GetGridSize(std::size_t vectSize) {
        return (vectSize + blockSize - 1) / blockSize;
    }

    __global__ void GpuAddVect(float* vect1, float* vect2, float* resultVect, std::size_t vectSize){
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < vectSize) resultVect[i] = vect1[i] + vect2[i];
    };

    void FullGpuAddVect(float* vect1, float* vect2, float* resultVect, std::size_t vectSize){
        std::size_t gridSize = GetGridSize(vectSize);
        GpuAddVect <<< blockSize, gridSize >>>(vect1, vect2, resultVect, vectSize);

    };

    float* CpuAddVect(float* vect1, float* vect2, float* resultVect, std::size_t vectSize){
        for(int i = 0; i < vectSize; i++){
            resultVect[i] = vect1[i] + vect2[i];
        }
        
        return resultVect;
    };

    float* RunGpu(float* vect1, float* vect2, float* resultVect, std::size_t vectSize){
        std::size_t gridSize = GetGridSize(vectSize);
        float *devVect1, *devVect2, *devResult;
        
        hipMalloc(&devVect1, sizeof(float) * vectSize);
        hipMalloc(&devVect2, sizeof(float) * vectSize);
        hipMalloc(&devResult, sizeof(float) * vectSize);

        hipMemcpy(devVect1, vect1, sizeof(float) * vectSize, hipMemcpyHostToDevice);
        hipMemcpy(devVect2, vect2, sizeof(float) * vectSize, hipMemcpyHostToDevice);
        hipMemcpy(devResult, resultVect, sizeof(float) * vectSize, hipMemcpyHostToDevice);

        GpuAddVect <<< blockSize, gridSize >>>(devVect1, devVect2, devResult, vectSize);

        hipMemcpy(resultVect, devResult, sizeof(float) * vectSize, hipMemcpyDeviceToHost);
        
        hipFree(devVect1);
        hipFree(devVect2);
        hipFree(devResult);
        return resultVect;
    };

};