#include "hip/hip_runtime.h"
#include <iostream>
#include "gpu_addVect.cuh"
#include <hip/hip_runtime.h>

namespace AddVect{

    __global__ void GpuAddVect(float* vect1, float* vect2, float* resultVect){
        int i = threadIdx.x;
        resultVect[i] = vect1[i] + vect2[i];
    };

    void FullGpuAddVect(float* vect1, float* vect2, float* resultVect, int gridSize, float* kernel_ms){
        float *devVect1, *devVect2, *devResult;
        hipEvent_t start, stop;

        hipMalloc((void**)&devVect1, sizeof(float) * gridSize);
        hipMalloc((void**)&devVect2, sizeof(float) * gridSize);
        hipMalloc((void**)&devResult, sizeof(float) * gridSize);

        hipMemcpy(devVect1, vect1, sizeof(float) * gridSize, hipMemcpyHostToDevice);
        hipMemcpy(devVect2, vect2, sizeof(float) * gridSize, hipMemcpyHostToDevice);
        hipMemcpy(devResult, resultVect, sizeof(float) * gridSize, hipMemcpyHostToDevice);
        
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        GpuAddVect <<< 1, gridSize >>>(devVect1, devVect2, devResult);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(kernel_ms, start, stop);

        hipMemcpy(resultVect, devResult, sizeof(float) * gridSize, hipMemcpyDeviceToHost);
        hipFree(devVect1);
        hipFree(devVect2);
        hipFree(devResult);
    };

    float* AddingVectors::CpuAddVect(float* vect1, float* vect2, float* resultVect, int gridSize){
        for(int i = 0; i < gridSize; i++){
            resultVect[i] = vect1[i] + vect2[i];
        }
        
        return resultVect;
    };

    float* AddingVectors::RunGpu(float* vect1, float* vect2, float* resultVect, int gridSize){
        float *devVect1, *devVect2, *devResult;

        hipMalloc((void**)&devVect1, sizeof(float) * gridSize);
        hipMalloc((void**)&devVect2, sizeof(float) * gridSize);
        hipMalloc((void**)&devResult, sizeof(float) * gridSize);

        hipMemcpy(devVect1, vect1, sizeof(float) * gridSize, hipMemcpyHostToDevice);
        hipMemcpy(devVect2, vect2, sizeof(float) * gridSize, hipMemcpyHostToDevice);
        hipMemcpy(devResult, resultVect, sizeof(float) * gridSize, hipMemcpyHostToDevice);

        GpuAddVect <<< 1, gridSize >>>(devVect1, devVect2, devResult);

        hipMemcpy(resultVect, devResult, sizeof(float) * gridSize, hipMemcpyDeviceToHost);
        
        hipFree(devVect1);
        hipFree(devVect2);
        hipFree(devResult);
        return resultVect;
    };

};