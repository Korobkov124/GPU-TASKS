#include "hip/hip_runtime.h"
#include <iostream>
#include <gpu_addVect.cuh>

namespace AddVect{

    __global__ void GpuAddVect(float* vect1, float* vect2, float* resultVect, int N){
        int i = threadIdx.x;
        resultVect[i] = vect1[i] + vect2[i];
    };

    float* AddingVectors::CpuAddVect(float* vect1, float* vect2, float* resultVect, int N){
        for(int i = 0; i < N; i++){
            resultVect[i] = vect1[i] + vect2[i];
        }
        
        return resultVect;
    };

    float* AddingVectors::RunGpu(float* vect1, float* vect2, float* resultVect, int N){
        float *devVect1, *devVect2, *devResult;

        hipMalloc((void**)&devVect1, sizeof(float) * N);
        hipMalloc((void**)&devVect2, sizeof(float) * N);
        hipMalloc((void**)&devResult, sizeof(float) * N);

        hipMemcpy(devVect1, vect1, sizeof(float) * N, hipMemcpyHostToDevice);
        hipMemcpy(devVect2, vect2, sizeof(float) * N, hipMemcpyHostToDevice);
        hipMemcpy(devResult, resultVect, sizeof(float) * N, hipMemcpyHostToDevice);

        GpuAddVect <<< 1, N >>>(devVect2, devVect2, resultVect, N);

        hipMemcpy(resultVect, devResult, sizeof(float), hipMemcpyDeviceToHost);

        
        hipFree(devVect2);
        hipFree(devVect2);
        hipFree(devResult);
        return resultVect;
    };

};